
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

struct IntSandwich {
  int beginning;
  int middle[1];
  int end;  
};

__global__ void access_offset_kernel(int offset) {
    struct IntSandwich devMem;
    devMem.beginning = 0; devMem.middle[0] = 0; devMem.end = 0;
#ifdef R
    volatile int i = devMem.middle[offset];
#elif W
    devMem.middle[offset] = 42;
    devMem.middle[offset] = devMem.middle[offset] * 2; // for unused warning
#endif
}

int main(int argc, char** argv) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s -o <offset>\n", argv[0]);
        abort();
    }

    int offset = 0;
    int c;
    while ((c = getopt(argc, argv, "o:")) != -1) {
        switch(c) {
        case 'o':
            offset = atoi(optarg);
            break;
        default:
            fprintf(stderr, "Usage: %s -o <offset>\n", argv[0]);
            abort();
        }
    } 
  
    access_offset_kernel<<<1,1>>>(offset);

    hipDeviceReset();
    return 0;
}
