#include "hip/hip_runtime.h"
#include <limits.h>
#include <stdio.h>

__global__ void oversized_alloc_kernel()
{
  char* devMem = (char*)malloc(ULONG_MAX);
  free(devMem);
}

int main()
{
  cudaThreadSetLimit(hipLimitMallocHeapSize, 128*sizeof(char));
  oversized_alloc_kernel<<<1, 1>>>();

  hipDeviceReset();
  return 0;
}
