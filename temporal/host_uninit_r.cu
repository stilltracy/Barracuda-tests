
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>

#define ALLOC_SIZE 1024

__global__ void simple_kernel(int *hostMem) {
    hostMem[0] = 42;
}

int main() {
    int *hostMem;

    hipMalloc((void**)&hostMem, ALLOC_SIZE*sizeof(int));

    simple_kernel<<<1, 1>>>(hostMem);

    hipFree(hostMem);
    
    hipDeviceReset();
    return 0;
}
