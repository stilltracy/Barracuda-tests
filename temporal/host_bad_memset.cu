
#include <hip/hip_runtime.h>
#include <stdio.h>

#define ALLOC_SIZE 1024

int main() {
    int *hostAllocMem;

    hipMalloc((void**)&hostAllocMem, ALLOC_SIZE*sizeof(int));
    hipMemset(hostAllocMem, 0, (ALLOC_SIZE+1)*sizeof(int));
    hipFree(hostAllocMem);
    
    hipDeviceReset();
    return 0;
}
