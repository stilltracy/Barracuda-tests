
#include <hip/hip_runtime.h>
#include <stdio.h>

#define ALLOC_SIZE 1024

__global__ void simple_kernel(int *hostMem) {
    hostMem[0] = 42;
}

int main() {
    int *hostMem;

    hipMalloc((void**)&hostMem, ALLOC_SIZE*sizeof(int));
    hipFree(hostMem);
    
    simple_kernel<<<1, 1>>>(hostMem);

    hipDeviceReset();
    return 0;
}
