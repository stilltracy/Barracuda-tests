
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>

int main() {
    int *hostMem;

    hipMalloc((void**)&hostMem, ULONG_MAX);
    hipFree(hostMem);
    
    hipDeviceReset();
    return 0;
}
