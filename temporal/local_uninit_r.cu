
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>

#define ALLOC_SIZE 1024

__global__ void simple_kernel() {
    int devMem[ALLOC_SIZE];
    int i = devMem[0];
    i = i*i;  // for unreferenced warning
}

int main() {
    simple_kernel<<<1, 1>>>();

    hipDeviceReset();
    return 0;
}
