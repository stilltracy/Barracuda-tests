
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SRC_SIZE 65536
#define DST_SIZE 65536
#define CPY_SIZE 8192

int main() {
    int *h_mem = (int*)malloc(SRC_SIZE*sizeof(int));

    int *d_mem;
    hipMalloc((void**)&d_mem, DST_SIZE*sizeof(int));

    hipMemcpy(d_mem, h_mem, CPY_SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(h_mem, d_mem, CPY_SIZE*sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(d_mem);
    free(h_mem);
    
    hipDeviceReset();
    return 0;
}
